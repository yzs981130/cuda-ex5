#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_WIDTH 16 
#define TILE_WIDTH  BLOCK_WIDTH

extern "C" void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int m, int p, int n);

__global__
void gpu_mat_mul_kernel(float* M, float* N, float* P, int m, int p, int n){

  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the P element to work on
  // Each thread works on an element of P
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  float sum = 0;
  int phase_num = p/TILE_WIDTH + 1;

  // Each thread loads 'Row'th row of M and 'Col'th column of N
  for (int ph = 0; ph < phase_num; ph++) {    

    // Collaboratively load data into shared memory
    //Mds[ty][tx] = M[Row * width + ph * TILE_WIDTH + tx];   
    //Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * width + Col];
    if ((ph * TILE_WIDTH + tx) < p && Row < m) {
      Mds[ty][tx] = M[Row * p + ph * TILE_WIDTH + tx];
    }
    else {
      Mds[ty][tx] = 0;
    }

    if ((ph * TILE_WIDTH + ty) < p && Col < n) {
      Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * n + Col];
    }
    else {
      Nds[ty][tx] = 0;
    }

    __syncthreads();
    if (Row < m && Col < n) {
      for (int k = 0; k < TILE_WIDTH; k++) { 
        sum += Mds[ty][k] * Nds[k][tx];
      }
    }
    __syncthreads();
  }
  if (Row < m && Col < n) {
    P[Row * n + Col] = sum;
  }  
}

void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int m, int p, int n) {
  float *d_M, *d_N, *d_P;

  size_t size_of_float = sizeof(float);
  size_t size_M = m * p * size_of_float;
  size_t size_N = p * n * size_of_float;
  size_t size_P = m * n * size_of_float;

  hipMalloc((void**)&d_M, size_M);
  hipMalloc((void**)&d_N, size_N);
  hipMalloc((void**)&d_P, size_P);
    
  hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float elapsed_time = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid_dim(n/BLOCK_WIDTH + 1, m/BLOCK_WIDTH + 1, 1);
  dim3 block_dim(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  gpu_mat_mul_kernel<<<grid_dim, block_dim>>>(d_M, d_N, d_P, m, p, n);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
  // Free device memory for M, N, P
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  hipEventElapsedTime(&elapsed_time, start, stop);
    
  printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
  printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
  printf("  kernel time: %.5f sec\n", elapsed_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


